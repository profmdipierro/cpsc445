#include "hip/hip_runtime.h"
#include <stdio.h>

__global__ void reduce_sum(float * da, int N) {
  int W = blockDim.x;
  int tid = threadIdx.x;
  for(int i=tid+W; i<N; i+=W)  da[tid] += da[i];
}

int main() {
  //INPUTS
  int N = 1000;
    
  int *ha = new int[N];
  int *da;
  hipMalloc((void **)&da, N*sizeof(int));

  // set problem input (b)
  for (int i = 0; i<N; ++i) {
    ha[i] = i*i;
  }
  
  hipMemcpy(da, ha, N*sizeof(int), hipMemcpyHostToDevice);

  int W = 10;
  reduce_sum<<<1,W>>>(da, N);
  hipDeviceSynchronize();

  int sums[10];
  hipMemcpy(sums, da, W*sizeof(int), hipMemcpyDeviceToHost);

  int sum=0;
  for(int i=0; i<W; i++) sum+=sums[i];
  
  printf("%i", sum);
  hipFree(da);
  free(ha);
  return 0;
}
