
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void f() {
  printf("Hello from %i, %i\n", blockIdx.x, threadIdx.x);
}

int main(void) {

f<<<5, 3>>>();
  hipDeviceSynchronize();

  f<<<7, 2>>>();
  hipDeviceSynchronize();

return 0;
}
