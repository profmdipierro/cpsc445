
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void reduce_sum(int * da, int N) {
  int W = blockDim.x;
  int stride = W * 2;
  int tid = threadIdx.x;
  for(int i=tid+stride; i<N; i+=stride) da[tid]+=da[i];
  __syncthreads();


  for(int delta=1; delta<=W; delta*=2) {
    int i = tid*(2*delta);
    if (i + delta < N) {
      da[i] += da[i+delta];
      printf("%i (%i): %i\n", i, delta, da[i]);
    }
    __syncthreads();
  }
}

int main() {
  //INPUTS
  int N = 40;

  int *ha = new int[N];
  int *da;
  hipMalloc((void **)&da, N*sizeof(int));

  // set problem input (b)
  for (int i = 0; i<N; ++i) {
    ha[i] = i*i;
  }
  
  hipMemcpy(da, ha, N*sizeof(int), hipMemcpyHostToDevice);

  int W = 16;
  reduce_sum<<<1,W>>>(da, N);
  hipDeviceSynchronize();

  int sum;
  hipMemcpy(&sum, da, sizeof(int), hipMemcpyDeviceToHost);

  int expected_sum = (N-1)*N*(2*N-1)/6;
  printf("%i (should be %i)", sum, expected_sum);
  hipFree(da);
  free(ha);
  return 0;
}
