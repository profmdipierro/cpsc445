
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void reduce_sum_step1(int * da, int N) {
  int B = gridDim.x;
  int W = blockDim.x;
  int shift = W * B;

  int gid = blockIdx.x * blockDim.x + threadIdx.x;
  
  for(int i=gid+shift; i<N; i+=shift) {
    da[gid]+=da[i];
  }
  
  __syncthreads();
  
  shift = blockDim.x * blockIdx.x;
  for(int delta=1; delta<W; delta*=2) {    
    int i = threadIdx.x;
    if (i + delta < W) {
      da[i+shift] += da[i+shift+delta];
    }
    __syncthreads();
  }  
}

__global__ void reduce_sum_step2(int * da, int W) {
  int B = blockDim.x;
  int shift = B;
  int tid = threadIdx.x;
  for(int delta=1; delta<B; delta*=2) {    
    int i = tid*2*delta;
    if (i + delta < B) {
      da[i*W] += da[(i+delta)*W];
    }
    __syncthreads();
  }  
}

int main() {
  //INPUTS
  int N = 1000;

  int *ha = new int[N];
  int *hb = new int[N];
  int *da;
  hipMalloc((void **)&da, N*sizeof(int));

  // set problem input (b)
  for (int i = 0; i<N; ++i) {
    ha[i] = i*i;
  }
  
  hipMemcpy(da, ha, N*sizeof(int), hipMemcpyHostToDevice);

  int B = 3;
  int W = 16;
  reduce_sum_step1<<<B,W>>>(da, N);
  hipDeviceSynchronize();
  reduce_sum_step2<<<1,B>>>(da, W);
  hipDeviceSynchronize();

  int sum;
  // cudaMemcpy(&sum, da, sizeof(int), cudaMemcpyDeviceToHost);
  hipMemcpy(hb, da, N*sizeof(int), hipMemcpyDeviceToHost);

  printf("%i\n", hb[0]);
  // printf("%i\n", hb[16]);
  // printf("%i\n", hb[32]);
  // printf("%i\n", hb[32]);

  sum = hb[0]; //  + hb[32];
  int expected_sum =  (N-1)*N*(2*N-1)/6;
  printf("%i (should be %i)", sum, expected_sum);
  hipFree(da);
  free(ha);
  free(hb);
  return 0;
}
