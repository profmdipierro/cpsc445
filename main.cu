
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void reduce_sum(int * da, int N) {
  int W = blockDim.x;
  int stride = W * 2;
  int tid = threadIdx.x;
  for(int i=tid+stride; i<N; i+=stride) da[tid]+=da[i];
  __syncthreads();
  for(int delta=1; delta<=W; delta*=2) {
    int i = tid*(2*delta);
    if (i + delta < N) {
      da[i] += da[i+delta];
    }
    __syncthreads();
  }
}

int main() {
  //INPUTS
  int N = 1000;
    
  int *ha = new int[N];
  int *da;
  hipMalloc((void **)&da, N*sizeof(int));

  // set problem input (b)
  for (int i = 0; i<N; ++i) {
    ha[i] = i*i;
  }
  
  hipMemcpy(da, ha, N*sizeof(int), hipMemcpyHostToDevice);

  int W = 16;
  reduce_sum<<<1,W>>>(da, N);
  hipDeviceSynchronize();

  int sum;
  hipMemcpy(&sum, da, sizeof(int), hipMemcpyDeviceToHost);

  printf("%i", sum);
  hipFree(da);
  free(ha);
  return 0;
}
