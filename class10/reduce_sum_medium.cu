
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void reduce_sum(int * da, int N) {
  int W = blockDim.x;
  int tid = threadIdx.x;
  for(int i=tid+W; i<N; i+=W) da[tid]+=da[i];
  __syncthreads();


  for(int delta=1; delta<W; delta*=2) {
    int i = tid*2*delta;
    if (i + delta < N) {
      da[i] += da[i+delta];
    }
    __syncthreads();
  }
}

int main() {
  //INPUTS
  int N = 1000;

  int *ha = new int[N];
  int *da;
  hipMalloc((void **)&da, N*sizeof(int));

  // set problem input (b)
  for (int i = 0; i<N; ++i) {
    ha[i] = i*i;
  }
  
  hipMemcpy(da, ha, N*sizeof(int), hipMemcpyHostToDevice);

  int W = 16;
  reduce_sum<<<1,W>>>(da, N);
  hipDeviceSynchronize();

  int sum;
  hipMemcpy(&sum, da, sizeof(int), hipMemcpyDeviceToHost);

  int expected_sum = (N-1)*N*(2*N-1)/6;
  printf("%i (should be %i)", sum, expected_sum);
  hipFree(da);
  free(ha);
  return 0;
}
